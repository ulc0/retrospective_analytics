/*
Kernels for attention forward pass.

Compile example:
nvcc -O3 --use_fast_math attention_forward.cu -o attention_forward

version 1 is naive port from CPU code to kernel, parallelize over batch, time, heads only
./attention_forward 1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void attention_forward_cpu(float* out, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int h = 0; h < NH; h++) {
                float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_bth = att + b*NH*T*T + h*T*T + t*T;

                // pass 1: calculate query dot key and maxval
                float maxval = -10000.0f; // TODO something better
                for (int t2 = 0; t2 <= t; t2++) {
                    float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

                    // (query_t) dot (key_t2)
                    float val = 0.0f;
                    for (int i = 0; i < hs; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }

                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 < T; t2++) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to PyTorch
                        att_bth[t2] = 0.0f;
                    }
                }

                // pass 4: accumulate weighted values into the output of attention
                float* out_bth = out + b * T * C + t * C + h * hs;
                for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
                for (int t2 = 0; t2 <= t; t2++) {
                    float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
                    float att_btht2 = att_bth[t2];
                    for (int i = 0; i < hs; i++) {
                        out_bth[i] += att_btht2 * value_t2[i];
                    }
                }
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void attention_query_key_kernel1(float* preatt, float* inp,
                                           int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) { return; } // autoregressive mask
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);

        int C3 = C*3;
        int hs = C / NH; // head size
        float* query_t = inp + b * T * C3 + t * C3 + h * hs;
        float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = 0; i < hs; i++) {
            val += query_t[i] * key_t2[i];
        }
        val *= 1.0 / sqrtf(hs);

        preatt[idx] = val;
    }
}

__global__ void attention_softmax_kernel1(float* att, float* preatt,
                                         int B, int T, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        // find maxval
        float maxval = -10000.0f; // TODO something better
        for (int t2 = 0; t2 <= t; t2++) {
            if (preatt_bth[t2] > maxval) {
                maxval = preatt_bth[t2];
            }
        }

        // calculate the exp and keep track of sum
        float expsum = 0.0f;
        for (int t2 = 0; t2 <= t; t2++) {
            float expv = expf(preatt_bth[t2] - maxval);
            expsum += expv;
            att_bth[t2] = expv;
        }
        float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

        // normalize to get the softmax
        for (int t2 = 0; t2 < T; t2++) {
            if (t2 <= t) {
                att_bth[t2] *= expsum_inv;
            } else {
                // causal attention mask. not strictly necessary to set to zero here
                // only doing this explicitly for debugging and checking to PyTorch
                att_bth[t2] = 0.0f;
            }
        }
    }
}

__global__ void attention_value_kernel1(float* out, float* att, float* inp,
                                       int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        int C3 = C*3;
        int hs = C / NH; // head size

        float* out_bth = out + b * T * C + t * C + h * hs;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
        for (int t2 = 0; t2 <= t; t2++) {
            float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            for (int i = 0; i < hs; i++) {
                out_bth[i] += att_btht2 * value_t2[i];
            }
        }
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void attention_forward1(float* out, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // attention calculation
    int total_threads = B * NH * T * T;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    attention_query_key_kernel1<<<num_blocks, block_size>>>(preatt, inp, B, T, C, NH);
    // softmax and value accumulation
    total_threads = B * T * NH;
    num_blocks = CEIL_DIV(total_threads, block_size);
    attention_softmax_kernel1<<<num_blocks, block_size>>>(att, preatt, B, T, NH);
    attention_value_kernel1<<<num_blocks, block_size>>>(out, att, inp, B, T, C, NH);
}

// kernel version dispatch
void attention_forward(int kernel_num,
                       float* out, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    switch (kernel_num) {
        case 1:
            attention_forward1(out, preatt, att, inp, B, T, C, NH, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int NH = 12;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* preatt = (float*)malloc(B * NH * T * T * sizeof(float));
    float* att = (float*)malloc(B * NH * T * T * sizeof(float));
    float* inp = make_random_float(B * T * 3 * C);

    // move to GPU
    float* d_out;
    float* d_preatt;
    float* d_att;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_preatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    attention_forward_cpu(out, preatt, att, inp, B, T, C, NH);
    attention_forward(kernel_num, d_out, d_preatt, d_att, d_inp, B, T, C, NH, 256);

    float* out_gpu = (float*)malloc(B * T * C * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * C; i++) {
        // print the first few comparisons
        if (i < 5) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-4) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match!\n");

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 10;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            attention_forward(kernel_num, d_out, d_preatt, d_att, d_inp, B, T, C, NH, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(out);
    free(preatt);
    free(att);
    free(inp);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_preatt));
    cudaCheck(hipFree(d_att));
    cudaCheck(hipFree(d_inp));

    return 0;
}